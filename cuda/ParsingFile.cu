#include <iostream>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>


using namespace std;

class Node {
    public:
        int symbol;
        string symbolStr;
        Node* left;
        Node* right;
        
        Node(int s)
        {
            this->symbol = s;
            this->left = NULL;
            this->right = NULL;
        }
        
        string listTree(string* nts, string* terminals, int &indexTerminals)
        {
            string list;
            
            list = "['" + nts[this->symbol] + "' ";
            
            if ( this->left != NULL )
            {
                list = list + ", ";
                list = list + (this->left)->listTree(nts, terminals, indexTerminals);
            }
            
            if ( this->right != NULL )
            {
                list = list + ", ";
                list = list + (this->right)->listTree(nts, terminals, indexTerminals);
            }
            
            if ( this->left == NULL && this->right == NULL ) 
            {
                list = list + string(", '") + terminals[indexTerminals] + string("'");
                indexTerminals = indexTerminals + 1;
            }
            
            list = list + "]";
            
            return list;
        }
};


bool fileExists(string filename)
{
    ifstream file(filename);
    bool exist = file.good();
    file.close();
    return exist;
}


// Verifica che 'symbol' sia presente nell'array dei simboli non terminali 'nts'
int getIndexNT(string symbol, string* nts, int nNT)
{
    int index = 0;
    string nt;
    bool found = false;
    while ( index < nNT && !found) 
    {
        nt = nts[index];
        //cout << "nonTerminals[" << index << "] = " << nt << "\n";
        if ( nt.compare(symbol) == 0 )
        {
            found = true;
        }
        else
        {
            index = index + 1;
        }
    }
    if ( !found )
    {
        index = -1;
    }
    return index;
}

// Funzione per dividere in un unico array la regola con posizione 0 leftside e le restanti due rightside
// Inutilizzata
// SI -> NP VP
// ai = 2,  lsi = 8
string* getSidesRule(string rule, int* numberRightSides)
{
    //cout << "Dentro la funzione per ottenere i dati della regola\n";
    int arrowIndex = 0;
    int lastSpaceIndex = 0;
    string arrow = " -> ";
    string space = " ";
    string rightSide1;
    string rightSide2;
    string leftSide;
    string* ruleSplitted;
    arrowIndex = rule.find(arrow);
    leftSide = rule.substr(0, arrowIndex);
    lastSpaceIndex = rule.rfind(space);
    if (lastSpaceIndex != arrowIndex+3)
    {
        //cout << "E' una regola binaria\n";
        // Se l'ultimo spazio non è quello dopo la freccia allora è una regola binaria
        (*numberRightSides) = 2;
        rightSide1 = rule.substr(arrowIndex+4, lastSpaceIndex-arrowIndex-4);
        rightSide2 = rule.substr(lastSpaceIndex+1);
    }
    else
    {
        //cout << "E' una regola unaria\n";
        // Altrimenti è una regola unaria
        (*numberRightSides) = 1;
        rightSide1 = rule.substr(arrowIndex+4);
    }
    
    //cout << "Numero di elementi a destra: " << (*numberRightSides) << "\n";
    
    ruleSplitted = new string[*(numberRightSides)+1];
    //cout << "Allocata l'array contenente gli elementi\n";
    if (*(numberRightSides) == 2)
    {
        ruleSplitted[0] = leftSide;
        ruleSplitted[1] = rightSide1;
        ruleSplitted[2] = rightSide2;
        //cout << "E' una regola binaria\n";
        //cout << "Regola sinistra: " << leftSide << "\n";
        //cout << "Regola destra prima: " << rightSide1 << "\n";
        //cout << "Regola destra seconda: " << rightSide2 << "\n";
        
    }
    else
    {
        ruleSplitted[0] = leftSide;
        ruleSplitted[1] = rightSide1;
        //cout << "E' una regola unaria\n";
        //cout << "Regola sinistra: " << leftSide << "\n";
        //cout << "Regola destra: " << rightSide1 << "\n";
    }
    return ruleSplitted;
}

__device__ void lock(int* mutex)
{
    /*printf("Lock!\n");
    while ( atomicCAS(mutex, 0, 1) != 0 );
    printf("Lockato con successo\n");*/
}

__device__ void unlock(int* mutex)
{
    /*printf("Unlock!\n");
    atomicExch(mutex, 0);
    printf("Unlockato con successo\n");*/
}



// Funzione kernel
__global__ void parsing(double* table, int* grammarsRules, double* grammarsProb, int nWords, int nGrammars, int nNT, int* back, int index, int split, int end)
{
    //printf("Entrato nel gpu thread\n");
    int leftSide;
    int rightSide1;
    int rightSide2;
    //int indexID = threadIdx.x;
    int indexID = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;
    double probSplitting;
    //pedix = 0;
    //int pedix = indexID;
    int pedix = 0;
    //cout << "Indice thread gpu: " << pedix << "\n";
    //printf("Indice thread gpu: %d \n", indexID);
    //printf("nWords, nGrammars, nNT -> %d %d %d \n", nWords, nGrammars, nNT);
    //while ( pedix < nGrammars )
    while ( pedix < nGrammars )
    {
        //printf("Sono nel while!\n");
        leftSide = grammarsRules[pedix*3];
        if ( leftSide == indexID )
        {
            rightSide1 = grammarsRules[(pedix*3)+1];
            rightSide2 = grammarsRules[(pedix*3)+2];
            //cout << "\t\t\t Regola sinistra: " << leftSide << "\n";
            //printf("%d: Regola: %d -> %d %d\n", indexID, leftSide, rightSide1, rightSide2);
            
            // Caso regola binaria
            if ( rightSide2 != -1 )
            {
                //lock(mutex);
                //cout << "\t\t\t E' una regola binaria " << leftSide << " -> " << rightSide1 << " " << rightSide2 << "\n";
                //printf("%d: E' una regola binaria %d -> %d %d \n", indexID, leftSide, rightSide1, rightSide2);
                //if ( table[index][split][rightSide1] != 0.0 && table[split+1][end][rightSide2] != 0.0 )
                //printf("Indici %d-%d-%d e %d-%d-%d  %d e %d e valori nella tabella del prossimo controllo: %f e %f \n", index, split, rightSide1, split+1, end, rightSide2, index + (nWords*split) + (nWords*nWords*rightSide1), split+1 + (nWords*end) + (nWords*nWords*rightSide2), table[index + (nWords*split) + (nWords*nWords*rightSide1)], table[split+1 + (nWords*end) + (nWords*nWords*rightSide2)]);
                if ( table[index + (nWords*split) + (nWords*nWords*rightSide1)] != 0.0 && table[split+1 + (nWords*end) + (nWords*nWords*rightSide2)] != 0.0 )
                {
                    //cout << "\t\t\t I simboli a destra hanno una probabilità diversa da zero \n";
                    //printf("\t\t\t I simboli a destra hanno una probabilità diversa da zero \n");
                    //probSplitting = grammarsProb[pedix] * table[index][split][rightSide1] * table[split+1][end][rightSide2];
                    probSplitting = grammarsProb[pedix] * table[index + (nWords*split) + (nWords*nWords*rightSide1)] * table[split+1 + (nWords*end) + (nWords*nWords*rightSide2)];
                    //if ( probSplitting > table[index][end][leftSide] )
                    if ( probSplitting > table[index + (end*nWords) + (leftSide*nWords*nWords)] )
                    {
                        //cout << "\t\t\t\t OK! della regola " << nonTerminals[leftSide] << " -> " << nonTerminals[rightSide1] << " " << nonTerminals[rightSide2] << "con probabilita: " << probSplitting << "\n";
                        //printf("\t\t\t\t %d : OK! della regola binaria %d -> %d %d \n", indexID, leftSide, rightSide1, rightSide2);
                        //table[index][end][leftSide] = probSplitting;
                        // Cambio con atomicMax?
                        table[index + (nWords*end) + (nWords*nWords*leftSide)] = probSplitting;
                        back[index + (end*nWords) + (nWords*nWords*leftSide) + (nNT*nWords*nWords*0)] = split;
                        back[index + (end*nWords) + (nWords*nWords*leftSide) + (nNT*nWords*nWords*1)] = rightSide1;
                        back[index + (end*nWords) + (nWords*nWords*leftSide) + (nNT*nWords*nWords*2)] = rightSide2;
                    }
                }
                //unlock(mutex);
            }
            // Caso regola unaria
            else if ( rightSide2 == -1 )
            {
                //indexRightSide1NT = getIndexNT(ruleSplitted[1], nonTerminals, nNT);
                //cout << "Indice nella tabella dei simboli non terminali del simbolo a destra: " << indexRightSide1NT << "\n";
                // Ignoro le regole unarie che portano ad un simbolo terminale perché le ho già studiate
                if ( rightSide1 != -1 && rightSide1 < nNT )
                {
                    //lock(mutex);
                    //cout << "\t\t\t E' una regola unaria senza simbolo terminale: " << nonTerminals[leftSide] << " -> " << nonTerminals[rightSide1] << "\n";
                    //printf("E' una regola unaria senza simbolo terminale: %d -> %d \n", leftSide, rightSide1);
                    //if ( table[index][split][rightSide1] != 0.0 )
                    //printf("Indice %d-%d-%d con indici totale %d e valore nella tabella nel prossimo controllo %f \n",index, split, rightSide1, index + (split*nWords) + (nWords*nWords*rightSide1), table[index + (split*nWords) + (nWords*nWords*rightSide1) ]);
                    if ( table[index + (split*nWords) + (nWords*nWords*rightSide1) ] != 0.0 )
                    {
                        //probSplitting = grammarsProb[pedix] * table[index][split][rightSide1];
                        probSplitting = grammarsProb[pedix] * table[index + (split*nWords) + (nWords*nWords*rightSide1)];
                        //if ( probSplitting > table[index][split][leftSide] )
                        if ( probSplitting > table[index + (split*nWords) + (nWords*nWords*leftSide) ] )
                        {
                            //cout << "\t\t\t\t OK! della regola unaria " << nonTerminals[leftSide] << " -> " << nonTerminals[rightSide1] << " con probabilita: " << probSplitting << "\n";
                            //printf("\t\t\t\t %d : OK! della regola unaria %d -> %d\n", indexID, leftSide, rightSide1);
                            //table[index][split][leftSide] = probSplitting;
                            table[index + (split*nWords) + (nWords*nWords*leftSide)] = probSplitting;
                            back[index + (split*nWords) + (nWords*nWords*leftSide) + (nNT*nWords*nWords*0)] = split;
                            back[index + (split*nWords) + (nWords*nWords*leftSide) + (nNT*nWords*nWords*1)] = rightSide1;
                            back[index + (split*nWords) + (nWords*nWords*leftSide) + (nNT*nWords*nWords*2)] = -1;
                            
                        }
                    }
                    //unlock(mutex);
                }
            }
        }
        //cout << "passo alla regola successiva\n";
        pedix = pedix + 1;
        
        //pedix = pedix + stride;
    }
}



// Array di parole con relativa dimensione
// Array delle regole delle grammatiche con relativa dimensione
// Array delle probabilità delle regole della grammatica
// Array dei simboli non terminali con relativa dimensione
// Matrice per ottenere l'albero passata per riferimento?
int* cykParsing(string* words, int nWords, int* grammarsRules, int nGrammars, double* grammarsProb, string* nonTerminals, int nNT, string* terminals, int nTerminals)
{
    //double*** table;
    double* table;
    int* back;
    // Memoria device
    double* tableDevice;
    int* backDevice;
    //int* nWordsDevice;
    //int* nGrammarsDevice;
    double* grammarsProbDevice;
    int* rulesDevice;
    int* mutex;
    int state = 0;
    
    //double*** back;
    int i;
    int j;
    int k;
    int index;
    int pedix;
    int numberRightSides;
    
    //string* ruleSplitted;
    int leftSide;
    int rightSide1;
    int rightSide2;
    
    //int* rule;
    double probSplitting;
    int length;
    int split;
    int end;
    // Iterazione per inizializzare la matrice contenente le probabilità delle grammatiche
    cout << "Cykparsing - creazione matrici\n";
    table = new double[nWords*nWords*nNT];
    back = new int[nWords*nWords*nNT*3];
    i = 0;
    while ( i < nWords*nWords*nNT )
    {
        table[i] = 0.0;
        i = i + 1;
    }
    
    i = 0;
    while ( i < nWords*nWords*nNT*3 )
    {
        back[i] = -1;
        i = i + 1;
    }
    

    cout << "Cykparsing - matrici create\n";
    // Inizializzazione matrici con probabilità delle regole X -> xi delle parole
    // index indice delle parole ; pedix indice delle grammatiche
    cout << "inizializzazione matrici con probabilità delle parole\n";
    index = 0;
    while ( index < nWords ) 
    {
        pedix = 0;
        while ( pedix < nGrammars )
        {
            //cout << "\t "<< index << ": studio regola " << rule << " " << rule[0] << " " << rule[1] << " " << rule[2] << "\n";
            //ruleSplitted = getSidesRule(rule, &numberRightSides);
            leftSide = grammarsRules[pedix*3];
            rightSide1 = grammarsRules[(pedix*3)+1];
            rightSide2 = grammarsRules[(pedix*3)+2];
            
            if ( rightSide2 == -1 && rightSide1 >= nNT && words[index].compare(terminals[rightSide1-nNT]) == 0 )
            {
                //table[index][index][leftSide] = grammarsProb[pedix];
                table[index + (nWords*index) + (leftSide*nWords*nWords)] = grammarsProb[pedix];
                //cout << "\tSettato nella tabella con indice " << index << "-" << index << "-" << leftSide << " e indice totale " << index + (nWords*index) + (leftSide*nWords*nWords) << " e regola: "<< nonTerminals[leftSide] << " -> " << terminals[rightSide1-nNT] << " con valore :" << grammarsProb[pedix] << "=" << table[index + (nWords*index) + (leftSide*nWords*nWords)] << "\n";
            }
            pedix = pedix + 1;
        }
        index = index + 1;
    }
    
    
     // Allocazione memoria device
    /*    // Memoria device
        double* tableDevice;
        int* nWordsDevice;
        int* nGrammarsDevice;
        double* grammarsProbDevice;*/
    hipMalloc((void**)&tableDevice, sizeof(double)*nWords*nWords*nNT);
    hipMalloc((void**)&backDevice, sizeof(int)*nWords*nWords*nNT*3);
    ///cudaMalloc((void**)&nWordsDevice, sizeof(int));
    ///cudaMalloc((void**)&nGrammarsDevice, sizeof(int));
    hipMalloc((void**)&grammarsProbDevice, sizeof(double)*nGrammars);
    hipMalloc((void**)&rulesDevice, sizeof(int)*nGrammars*3);
    //cudaMalloc((void**)&mutex, sizeof(int));
    
    
    // Copia dati da memoria host a memoria device
    hipMemcpy(tableDevice, table, sizeof(double)*nWords*nWords*nNT, hipMemcpyHostToDevice);
    //cudaMemcpy(nWordsDevice, &nWords, sizeof(int), cudaMemcpyHostToDevice);
    //cudaMemcpy(nGrammarsDevice, &nGrammars, sizeof(int), cudaMemcpyHostToDevice);
    hipMemcpy(grammarsProbDevice, grammarsProb, sizeof(double)*nGrammars, hipMemcpyHostToDevice);
    hipMemcpy(rulesDevice, grammarsRules, sizeof(int)*nGrammars*3, hipMemcpyHostToDevice); 
    //cudaMemcpy(mutex, &state, sizeof(int), cudaMemcpyHostToDevice);
    hipMemcpy(backDevice, back, sizeof(int)*nWords*nWords*nNT*3, hipMemcpyHostToDevice);
    
    cout << "Dimensioni nWords, nGrammars, nNT : " << nWords << " " << nGrammars << " " << nNT;
    
    length = 1;
    // index, split, end per la matrice
    // pedix per le grammatiche
    while ( length < nWords )
    {
        index = 0;
        while ( index < nWords - length )
        {
            end = index + length;
            split = index;
            while ( split < end ) 
            {   
                ///cout << "\t Studio la regola binaria con sottostringhe pari a : " << index << "-" << split << " e " << split+1 << "-" << end << "\n";
                //parsing(double* table, int* grammarsRules, double* grammarsProb, int nWords, int nGrammars, int nNT,  int index, int split, int end)
                // block_size = 256 grid_size = N+block_size  / block_size
                parsing<<<8,256>>>(tableDevice, rulesDevice, grammarsProbDevice, nWords, nGrammars, nNT, backDevice, index, split, end);
                hipDeviceSynchronize();
                split = split + 1;
            } 
            index = index + 1;
        }
        length = length + 1;
    }
    //cout << "Dopo i gpu threading\n";
    
    int indexStartSymbol;
    string startSymbol = "S";
    indexStartSymbol = getIndexNT(startSymbol, nonTerminals, nNT);
    
    //cout << "Ottenuto il simbolo di inizio frase " << indexStartSymbol << "\n";
    
    // Trasferisco i dati dalla memoria device alla memoria host
    hipMemcpy(table, tableDevice, sizeof(double)*nWords*nWords*nNT, hipMemcpyDeviceToHost);
    hipMemcpy(back, backDevice, sizeof(int)*nWords*nWords*nNT*3, hipMemcpyDeviceToHost);
    
    cout << "------PROBABILITA PARSING : " << table[0 + (nWords*(nWords-1)) + (indexStartSymbol*nWords*nWords)] << "\n";
    hipFree(tableDevice);
    hipFree(grammarsProbDevice);
    hipFree(rulesDevice);
    hipFree(backDevice);
    //cudaFree(mutex);
    //return table[0 + (nWords*(nWords-1)) + (indexStartSymbol*nWords*nWords)];
    if ( table[0 + (nWords*(nWords-1)) + (indexStartSymbol*nWords*nWords)] == 0.0 )
    {
        cout << "Tabella == 0 cancello\n";
        delete[] back;
        back = NULL;
    }
    delete[] table;
    return back;
}




// Procedura per leggere la grammatica dal file, nei parametri passa l'array delle regole e delle probabilità
int readGrammar(ifstream &file, int* &rules, double* &probs)
{
    string line;
    int nLines = 0;
    int index;
    
    int indexTab;
    int indexFirstSpace;
    int indexSecondSpace;
    
    int firstSymbol;
    int secondSymbol;
    int thirdSymbol;
    
    double prob;
    
    // Verifico quante regole ci sono nel file per allocare la memoria
    if ( file.is_open() )
    {
        // Prima riga è il numero di grammatiche
        getline(file, line);
        nLines = atoi(line.c_str());
        
        // Allocazione in memoria degli array per memorizzare i dati
        rules = new int[nLines*3];
        probs = new double[nLines];
        index = 0;
        while ( index < nLines )
        {
            rules[index*3] = 0;
            rules[(index*3)+1] = 0;
            rules[(index*3)+2] = 0;
            probs[index] = 0.0;
            index = index + 1;
        }
        
        index = 0;
        while ( getline(file, line) )
        {
            // La riga è composta da int int int[tabulazione]double
            indexTab = line.find("\t");
            // Dall'indice della tabulazione a fine riga c'è il double
            indexFirstSpace = line.find(" ");
            indexSecondSpace = line.find(" ", indexFirstSpace+1);
            
            firstSymbol = atoi(line.substr(0, indexFirstSpace).c_str());
            secondSymbol = atoi(line.substr(indexFirstSpace+1, indexSecondSpace-indexFirstSpace).c_str());
            thirdSymbol = atoi(line.substr(indexSecondSpace+1, indexTab-indexSecondSpace).c_str());
            //cout << "Il terzo simbolo della grammatica e' : " << thirdSymbol << "\n";
            prob = atof(line.substr(indexTab+1).c_str());
            
            rules[index*3] = firstSymbol;
            rules[(index*3)+1] = secondSymbol;
            rules[(index*3)+2] = thirdSymbol;
            
            //cout << "Il terzo simbolo della grammatica e' : " << thirdSymbol << " = " << rules[(index+3)+2] << " \n";
            
            probs[index] = prob;
            
            index = index + 1;
        }
        
        file.close();
    }
    else
    {
        cout << "Non è stato possibile aprire il file\n";
    }
    return nLines;
}

int readNonTerminals(ifstream &file, string* &nonTerminals)
{
    string line;
    int nLines;
    int index; 
    
    if ( file.is_open() )
    {
        // Ottengo il numero di grammatiche presenti nel file
        getline(file, line);
        nLines = atoi(line.c_str());
        
        // Allocazione memoria
        nonTerminals = new string[nLines];
        
        index = 0;
        while ( getline(file, line) )
        {
            //cout << "secondo giro non terminale = " << line << "\n";
            nonTerminals[index] = line;
            index = index + 1;
        }
        //cout << "Numero di non terminali trovati : " << nLines << "\n";
        file.close();
    }
    else
    {
        cout << "Errore durante l'apertura del file\n";
    }
    return nLines;
}

int* joinArray(int* array1, int* array2, int size1, int size2)
{
    int* total = new int[(size1+size2)*3];
    int index;
    int pedix;
    
    index = 0;
    pedix = 0;
    cout << "Inserisco l'array2 in total\n";
    while ( pedix < size2 )
    {
        //cout << "\t " << index << " = " << pedix << "\n";
        total[(index*3)] = array2[(pedix*3)];
        total[(index*3)+1] = array2[(pedix*3)+1];
        total[(index*3)+2] = array2[(pedix*3)+2];
        //cout << "\t " << total[index*3] << " = " << array2[pedix*3] << "\n";
        index = index + 1;
        pedix = pedix + 1;
    }
    cout << "Finito l'array2 ora l'indice è: " << index << "\n";
    cout << "Inserisco l'array1 in total\n";
    pedix = 0;
    while ( pedix < size1 )
    {
        //cout << "\t " << index*3 << " = " << pedix*3 << "\n";
        total[(index*3)] = array1[(pedix*3)];
        total[(index*3)+1] = array1[(pedix*3)+1];
        total[(index*3)+2] = array1[(pedix*3)+2];
        //cout << "\t " << total[(index*3)+2] << " = " << array1[(pedix*3)+2] << "\n";
        index = index + 1;
        pedix = pedix + 1;
    }
    cout << "Dimensioni array: " << size1 << " " << size2 << "\n";
    return total;
}

double* joinArray(double* array1, double* array2, int size1, int size2)
{
    double* total = new double[size1+size2];
    int index;
    int pedix;
    
    index = 0;
    pedix = 0;
    cout << "Inserisco l'array2 in total\n";
    while ( pedix < size2 )
    {
        //cout << "\t " << index << " = " << pedix << "\n";
        total[index] = array2[pedix];
        //cout << "\t " << total[index] << " = " << array2[pedix] << "\n";
        index = index + 1;
        pedix = pedix + 1;
    }
    
    pedix = 0;
    while ( pedix < size1 )
    {
        //cout << "\t " << index << " = " << pedix << "\n";
        total[index] = array1[pedix];
        //cout << "\t " << total[index] << " = " << array1[pedix] << "\n";
        index = index + 1;
        pedix = pedix + 1;
    }
    return total;
}

int readWords(ifstream &file, string* &words)
{
    string line;
    int nWords;
    int index;
    int start;
    int end;
    // La prima riga contiene il numero di parole della frase
    getline(file, line);
    nWords = atoi(line.c_str());
    
    words = new string[nWords];
    getline(file, line);
    
    index = 0;
    start = 0;    
    while ( index < nWords )
    {
        end = line.find("\t", start+1);
        if ( end != -1 )
        {
            words[index] = line.substr(start, end-start);
            start = end + 1;
        }
        else
        {
            // Ultima parola
            words[index] = line.substr(start);
        }
        index = index + 1;
    }
    return nWords;
}

int readTerminals(ifstream &file, string* &terminals)
{
    string line;
    int nTerminals;
    int index;
    int start;
    int end;
    // La prima riga contiene il numero di simboli terminali
    getline(file, line);
    nTerminals = atoi(line.c_str());
    
    terminals = new string[nTerminals];
    getline(file, line);
    
    index = 0;
    start = 0;    
    while ( index < nTerminals )
    {
        end = line.find("\t", start+1);
        if ( end != -1 )
        {
            terminals[index] = line.substr(start, end-start);
            start = end + 1;
        }
        else
        {
            // Ultima parola
            terminals[index] = line.substr(start);
        }
        index = index + 1;
    }
    return nTerminals;
}

void printTree(Node* tree, int level, string* nts, string* terminals, int &indexTerminals)
{
    int index = 0;
    while ( index < level )
    {
        cout << "\t";
        index = index + 1;
    }
    
    //cout << "[";
    cout << nts[tree->symbol];
    if ( tree->left != NULL )
    {
        printTree(tree->left, level+1, nts, terminals, indexTerminals);
    }

    
    if ( tree->right != NULL )
    {
        printTree(tree->right, level+1, nts, terminals, indexTerminals);
    }
    
    if ( tree->left == NULL && tree->right == NULL )
    {
        // Terminale
        cout << "\t" << terminals[indexTerminals];
        indexTerminals = indexTerminals + 1;
    }
    cout << "\n";
    //cout << "]";
}

// Inizio con start = 0, end = nWords-1, symbol = 'S' (indice)
Node* getTree(int* back, int nWords, int nNT, int start, int end, int symbol)
{
    int split;
    int left;
    int right;
    Node* tree = new Node(symbol);
    //cout << "Il nodo ha valore: " << symbol << "\n";
    split = back[start + (end*nWords) + (nWords*nWords*symbol) + (nNT*nWords*nWords*0)];
    if  ( split != -1 )
    {
        // Verifica se è ha un unico figlio o due 
        left = back[start + (end*nWords) + (nWords*nWords*symbol) + (nNT*nWords*nWords*1)];
        right = back[start + (end*nWords) + (nWords*nWords*symbol) + (nNT*nWords*nWords*2)];
        
        tree->left = getTree(back, nWords, nNT, start, split, left);
        // Se era una regola binaria ha anche il secondo figlio
        if ( right != -1 )
        {
            tree->right = getTree(back, nWords, nNT, split+1, end, right);
        }
    }

    return tree;
}


void divideLineWords(string line, string* words, int* tags, int nWords)
{
    
}

/*int readNews(ifstream fileNews, string** newsTitles, int** tagTitles, string** newsSubtitles, int** tagSubtitles , string** newsCorpus, int** tagCorpus)
{
    string line;
    string title;
    string subtitle;
    string corpus;
    int nNews;
    int index;
    int nWordsTitle;
    int nWordsSubtitle;
    int nWordsCorpus;
    
    
    if ( file.is_open() )
    {
        // La prima riga contiene il numero di notizie totali
        getline(file, line);
        nNews = atoi(line.c_str());
        
        // Allocazione matrice per contenere le parole e i tag
        newsTitles = new string*[nNews];
        newsSubtitles = new string*[nNews];
        newsCorpus = new string*[nNews];
        tagTitles = new int*[nNews];
        tagSubtitles = new int*[nNews];
        tagCorpus = new int*[nNews];
        
        index = 0;
        while ( index < nNews )
        {
            // Ogni notizia ha 6 righe: 3 categorie con 2 righe per numero parole e parole con i tag
            getline(file, line);
            nWordsTitle = atoi(nWordsTitle.c_str());
            newsTitles[index] = new string[nWordsTitle];
            tagTitles[index] = new int[nWordsTitle];
            getline(file, title);
            
            getline(file, line);
            nWordsSubtitle = atoi(nWordsSubtitle.c_str());
            newsSubtitles = new string[nWordsSubtitle];
            tagSubtitles = new int[nWordsSubtitle];
            getline(file, subtitle);
            
            getline(file, line);
            nWordsCorpus = atoi(nWordsCorpus.c_str());
            newsCorpus = new string[nWordsCorpus];
            tagCorpus = new int[nWordsCorpus];
            
            // parola tag \t parola ...
            // Ottengo le parole e i tag 
            index = index + 1;
        }
    }
    else
    {
        cout << "Errore durante l'apertura del file contenente le notizie\n";
    }
}*/

// Manca la procedura per ottenere l'albero
// L'albero array di oggetti?
// Le grammatiche nel file di testo divise da una tabulazione e \n
// 

string getNameCategory(int category)
{
    string cat;
    if ( category == 0 )
    {
        cat = "title";
    }
    if ( category == 1 )
    {
        cat = "subtitle";
    }
    if ( category == 2 )
    {
        cat = "corpus";
    }       
    return cat;
}


int main(int na, char **va)
{   
    // Array per le regole, probabilita, parole, simboli non terminali e simboli terminali
    int* rules;
    double* probs;
    string* words;
    string* nts;
    string* terminals;
    int* rulesTerminals;
    double* probsTerminals;
    int* rulesTotal;
    double* probsTotal;
    // Relative dimensioni
    int nGrammars;
    int nNT;
    int nWords;
    int nTerminalsGrammars;
    int nTerminals;
    int nTotalGrammars;
    // Variabili per navigare attraverso le notizie categorie e frasi
    int nNews;
    int category;
    int index;
    string* text;
    
    int v;
    int* parsed;
    string startSymbol = "S";
    int indexParsing = 0;
    // indice frasi e stringhe per passaggio da intero a stringa
    int indexSentence;
    char strIndexSentence[6];
    char strIndexNews[6];
    // Variabili per aprire i file
    string pathSentence;
    string pathParsed;
    string treeParsed;
    
    ifstream fileSentence;
    ifstream fileNT;
    ifstream fileGrammars;
    ofstream fileParsed;
    
    fileNT.open("politifact.nt.txt");
    fileGrammars.open("general.grammars.txt");
    
    cout << "Ottengo grammatiche di base\n";
    nGrammars = readGrammar(fileGrammars, rules, probs);
    cout << "Grammatiche di base lette\n";
    
    v = 0;
    while ( v < nGrammars ) 
    {
        //cout << "rules[" << v << "] = " << rules[v*3] << " " << rules[(v*3)+1] << " " << rules[(v*3)+2] << "\n";
        v = v + 1;
    }
    
    cout << "Ottengo i simboli non terminali\n";
    nNT = readNonTerminals(fileNT, nts);
    cout << "Simboli non terminali ottenuti\n";
    
    // Costante politifact
    nNews = 947;
    // Iterazione per effettuare il parsing delle frasi
    index = 295;
    while ( index <= nNews )
    {
        cout << "Prendo in considerazione la notizia " << index << "\n";
        category = 0;
        // Numero di categoria indica rispettivamente titolo sottotitolo e corpus
        while ( category < 3 )
        {
            cout << "\t con categoria " << category << "\n";
            indexSentence = 0;
            while ( indexSentence != -1 )
            {
                cout << "\t\t frase " << indexSentence << "\n";
                sprintf(strIndexSentence, "%d", indexSentence);
                sprintf(strIndexNews, "%d", index);
                
                pathSentence = "politifact/"+string(strIndexNews);
                pathSentence = pathSentence + "."+getNameCategory(category)+".";
                pathSentence = pathSentence + string(strIndexSentence)+".txt";
                
                pathParsed = "parsed/politifact/"+string(strIndexNews);
                pathParsed = pathParsed + "."+getNameCategory(category)+".";
                pathParsed = pathParsed + string(strIndexSentence)+".parsed";
    
                fileSentence.open(pathSentence);
                
                if ( fileSentence.is_open() )
                {
                    if ( !fileExists(pathParsed) )
                    {
                        // Prime due righe relative al numero di parole
                        //cout << "Leggo le parole della frase\n";
                        nWords = readWords(fileSentence, words);
                        // Segmentation fault se il numero di parole è troppo alto
                        if ( nWords <= 100 )
                        {
                            //cout << "Lette le parole \n";
                            // altre due righe relative al numero di simboli non terminali
                            // cout << "Leggo i simboli terminali della frase \n";
                            nTerminals = readTerminals(fileSentence, terminals);
                            //cout << "Letti i simboli terminali della frase \n";
                            // Il resto per le grammatiche
                            //cout << "Leggo le grammatiche dei simboli terminali della frase \n";
                            nTerminalsGrammars = readGrammar(fileSentence, rulesTerminals, probsTerminals);
                            //cout << "Lette le grammatiche dei simboli terminali della frase \n";
                            // Istanze per le grammatiche totali
                            //cout << "Unisco le regole \n";
                            rulesTotal = joinArray(rules, rulesTerminals, nGrammars, nTerminalsGrammars);
                            //cout << "Finito di unire le regole\n";
                            //cout << "Unisco le probabilita \n";
                            probsTotal = joinArray(probs, probsTerminals, nGrammars, nTerminalsGrammars);
                            //cout << "Finito di unire le probabiltia \n";
                            nTotalGrammars = nGrammars+nTerminalsGrammars;
                          
                            v = 0;
                            while ( v < nWords )
                            {
                                //cout << "words[" << v << "] = " << words[v] << "\n";
                                v = v + 1;
                            }
                            
                            v = 0;
                            while ( v < nTerminals )
                            {
                                //cout << "terminals[" << v << "] = " << terminals[v] << "\n";
                                v = v + 1;
                            }
                            
                            v = 0;
                            while ( v < nNT )
                            {
                                //cout << "nonTerminals[" << v << "] = " << nts[v] << "\n";
                                v = v + 1;
                            }
                            
                            v = 0;
                            while ( v < nTotalGrammars ) 
                            {
                                //cout << "totalRules[" << v << "] = " << rulesTotal[v*3] << " " << rulesTotal[(v*3)+1] << " " << rulesTotal[(v*3)+2] << "\n";
                                v = v + 1;
                            }
                            
                            // Esecuzione del parsing
                            cout << "Inizio parsing! "<< pathSentence << "\n";
                            parsed = cykParsing(words, nWords, rulesTotal, nTotalGrammars, probsTotal, nts, nNT, terminals, nTerminals);
                            if ( parsed != NULL )
                            {                    
                                cout << "Percorso file parsato: " << pathParsed << "\n";
                                fileParsed.open(pathParsed);
                                //cout << "Aperto il file \n";
                                indexParsing = 0;
                                treeParsed = getTree(parsed, nWords, nNT, 0, nWords-1, getIndexNT(startSymbol, nts, nNT))->listTree(nts, words, indexParsing);
                                cout << treeParsed << "\n";
                                fileParsed << treeParsed;
                                fileParsed.close();
                                
                                delete[] parsed;
                            }
                            else
                            {
                                fileParsed.open(pathParsed);
                                fileParsed << "No parsing";
                                fileParsed.close();
                            }
                            // Deallocazione e passaggio alla frase successiva
                            cout << "Deallocazione memoria\n";
                            delete[] rulesTotal;       
                            delete[] probsTotal;
                            cout << "Deallocazione terminali\n";
                            delete[] terminals;
                            cout << "Deallocazione roba terminale\n";
                            delete[] rulesTerminals;
                            delete[] probsTerminals;
                        }
                        else
                        {
                            cout << "Frase troppo lunga: " << nWords << "\n";
                            fileParsed.open(pathParsed);
                            fileParsed << "Lunga";
                            fileParsed.close();
                        }
                        // Deallocazione e passaggio alla frase successiva
                        //cout << "Deallocazione memoria\n";
                        // segfault
                        //delete rulesTotal;

                        cout << "Deallocazione words terminals\n";
                        delete[] words;
                    }
                    else
                    {
                        cout << "File è stato già parsato: " << pathParsed << "\n";
                    }
                                            
                    cout << "Chiudo il file contenente la notizia \n";
                    // Salvare il parsing da qualche parte
                    
                    fileSentence.close();
                    indexSentence = indexSentence + 1;
                    cout << "Passo alla frase successiva\n";
                }
                else
                {
                    cout << "Non esiste il file " << pathSentence << "\n";
                    // Passo alla categoria successiva visto che il file non esiste
                    category = category + 1;
                    indexSentence = -1;
                }
            }
        }
        index = index +1;
    }
    fileNT.close();
    fileGrammars.close();
    delete nts;
    delete probs;
    delete rules;
    //cykParsing(String* words, int nWords, String* grammarsRules, int nGrammars, double* grammarsProb, String* nonTerminals, int nNT)
    //prob = cykParsing(words, nWords, rules, nRules, probs, nts, nNT, terminals, nTerminals);
}